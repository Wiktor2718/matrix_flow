#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>

// config
#define SIZE 32
//#define NDEBUG
#define value_type float

typedef struct Matrix {
    value_type *data;
    size_t rows;
    size_t cols;
    bool row_major;
} Matrix;

#define element_wise_kernel_flat(name, operator)                                                        \
    extern "C" __global__ void name(value_type *res, value_type *a, value_type *b, size_t len) {        \
        int index = blockIdx.x * blockDim.x + threadIdx.x;                                              \
        if (index < len) res[index] = a[index] operator b[index];                                       \
    }

#define element_wise_kernel_t_right(name, operator)                                                     \
    extern "C" __global__ void name(value_type *res, value_type *a, value_type *b,                      \
    size_t rows, size_t cols, size_t len) {                                                             \
        int idx = blockIdx.x * blockDim.x + threadIdx.x;                                                \
        if (idx < len) {                                                                                \
            size_t index = (idx * cols) % (rows * cols) + idx / rows;                                   \
            assert(index < len);                                                                        \
            res[idx] = a[idx] operator b[index];                                                        \
        }                                                                                               \
    }

#define element_wise_kernel_t_left(name, operator)                                                      \
    extern "C" __global__ void name(value_type *res, value_type *a, value_type *b,                      \
    size_t rows, size_t cols, size_t len) {                                                             \
        int idx = blockIdx.x * blockDim.x + threadIdx.x;                                                \
        if (idx < len) {                                                                                \
            size_t index = (idx * cols) % (rows * cols) + idx / rows;                                   \
            assert(index < len);                                                                        \
            res[idx] = a[index] operator b[idx];                                                        \
        }                                                                                               \
    }

#define element_wise_operation(name, flat, transpose)                                                   \
    extern "C" void name(Matrix res, Matrix a, Matrix b, size_t len) {                                  \
        int block_size = SIZE*SIZE;                                                                     \
        int grid_size = (len + block_size - 1) / block_size;                                            \
        if (a.row_major == b.row_major) {                                                               \
            flat<<<grid_size, block_size>>>(res.data, a.data, b.data, len);                             \
        } else {                                                                                        \
            transpose<<<grid_size, block_size>>>(res.data, a.data, b.data, b.rows, b.cols, len);        \
        }                                                                                               \
    }

element_wise_kernel_flat(_matrix_add_flat, +)
element_wise_kernel_flat(_matrix_sub_flat, -)
element_wise_kernel_flat(_matrix_mul_flat, *)
element_wise_kernel_flat(_matrix_div_flat, /)

element_wise_kernel_t_right(_matrix_add_t_right, +)
element_wise_kernel_t_right(_matrix_sub_t_right, -)
element_wise_kernel_t_right(_matrix_mul_t_right, *)
element_wise_kernel_t_right(_matrix_div_t_right, /)

element_wise_kernel_t_left(_matrix_sub_t_left, -)
element_wise_kernel_t_left(_matrix_div_t_left, /)

element_wise_operation(matrix_add_t_right, _matrix_add_flat, _matrix_add_t_right)
element_wise_operation(matrix_sub_t_right, _matrix_sub_flat, _matrix_sub_t_right)
element_wise_operation(matrix_mul_t_right, _matrix_mul_flat, _matrix_mul_t_right)
element_wise_operation(matrix_div_t_right, _matrix_div_flat, _matrix_div_t_right)

element_wise_operation(matrix_sub_t_left, _matrix_sub_flat, _matrix_sub_t_left)
element_wise_operation(matrix_div_t_left, _matrix_div_flat, _matrix_div_t_left)

// matrix scalar operations
#define with_scalar_kernel(name, operator)                                                              \
    __global__ void name(value_type *res, value_type *a, value_type b, size_t len) {                    \
            int index = blockIdx.x * blockDim.x + threadIdx.x;                                          \
            if (index < len) res[index] = a[index] operator b;                                          \
    }

#define with_scalar_operation(name, kernel)                                                             \
    extern "C" void name(value_type *res, value_type *a, value_type b, size_t len) {                    \
        int block_size = SIZE*SIZE;                                                                     \
        int grid_size = (len + block_size - 1) / block_size;                                            \
        kernel<<<grid_size, block_size>>>(res, a, b, len);                                              \
    }

with_scalar_kernel(_matrix_scalar_add, +)
with_scalar_kernel(_matrix_scalar_sub, -)
with_scalar_kernel(_matrix_scalar_mul, *)
with_scalar_kernel(_matrix_scalar_div, /)

with_scalar_operation(matrix_scalar_add, _matrix_scalar_add)
with_scalar_operation(matrix_scalar_sub, _matrix_scalar_sub)
with_scalar_operation(matrix_scalar_mul, _matrix_scalar_mul)
with_scalar_operation(matrix_scalar_div, _matrix_scalar_div)

// scalar matrix operations
#define with_matrix_kernel(name, operator)                                                              \
    __global__ void name(value_type *res, value_type a, value_type *b, size_t len) {                    \
            int index = blockIdx.x * blockDim.x + threadIdx.x;                                          \
            if (index < len) res[index] = a operator b[index];                                          \
    }

#define with_matrix_operation(name, kernel)                                                             \
    extern "C" void name(value_type *res, value_type a, value_type *b, size_t len)  {                   \
        int block_size = SIZE*SIZE;                                                                     \
        int grid_size = (len + block_size - 1) / block_size;                                            \
        kernel<<<grid_size, block_size>>>(res, a, b, len);                                              \
    }

with_matrix_kernel(_scalar_matrix_sub, -)
with_matrix_kernel(_scalar_matrix_div, /)

with_matrix_operation(scalar_matrix_sub, _scalar_matrix_sub);
with_matrix_operation(scalar_matrix_div, _scalar_matrix_div);

// matrix operations
#define apply_kernel(name, transformation)                                                              \
    __global__ void name(value_type *res, value_type *matrix, size_t size) {                            \
        int index = blockIdx.x * blockDim.x + threadIdx.x;                                              \
        if (index < size) res[index] = transformation(matrix[index]);                                   \
    }

#define apply(name, kernel)                                                                             \
    extern "C" void name(value_type *res, value_type *matrix, size_t size) {                            \
        int block_size = SIZE*SIZE;                                                                     \
        int grid_size = (size + block_size - 1) / block_size;                                           \
        kernel<<<grid_size, block_size>>>(res, matrix, size);                                           \
    }

#define Sqware(x) ((x) * (x))
apply_kernel(_sqware, Sqware)
apply(sqware, _sqware)

#define Neg(x) -(x)
apply_kernel(_matrix_neg, Neg)
apply(matrix_neg, _matrix_neg)

// activations
#define reLu(x) ( ((x) > 0) ? (x) : ((x) * 0.01) )
#define reLuP(x) ( ((x) > 0) ? 1. : .01 )

apply_kernel(_relu, reLu)
apply(relu, _relu)
apply_kernel(_relu_prime, reLuP)
apply(relu_prime, _relu_prime)

// Perform matrix multiplication using cuBLAS
extern "C" void dot(Matrix res, Matrix left, Matrix right) {
    // Check for dimension mismatch
    if (left.cols != right.rows) {
        fprintf(stderr, "Matrix dimension mismatch: cannot multiply\n");
        exit(1);
    }

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Set parameters for cuBLAS
    const value_type alpha = 1.0f;
    const value_type beta = 0.0f;

    // cuBLAS expects column-major matrices by default, so if row_major is true, we need to transpose the matrices.
    hipblasOperation_t left_op = left.row_major ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t right_op = right.row_major ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    hipblasSgemm(
        handle,
        left_op,          // Transpose left if necessary
        right_op,         // Transpose right if necessary
        (int)left.rows,   // Number of rows in matrix A and C
        (int)right.cols,  // Number of columns in matrix B and C
        (int)left.cols,   // Number of columns in matrix A and rows in matrix B
        &alpha,           // Scaling factor for the product
        left.data,        // Matrix A
        (int)left.rows,   // Leading dimension of A
        right.data,       // Matrix B
        (int)right.rows,  // Leading dimension of B
        &beta,            // Scaling factor for C
        res.data,         // Result matrix C
        (int)res.rows     // Leading dimension of C
    );

    // Clean up cuBLAS handle
    hipblasDestroy(handle);
}